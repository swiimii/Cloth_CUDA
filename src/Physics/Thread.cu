#include <Cloth/Physics.cuh>

void* physicsThreadFunc(void* nothing) {
	// Initialize CUDA context
	DeviceData deviceData_host = {
		.particleCount = particleCount,
		.read = NULL,
		.write = NULL
	};
	DeviceData* deviceData_dev;

	hipMalloc(&deviceData_host.read, particleCount * sizeof(Particle));
	hipMalloc(&deviceData_host.write, particleCount * sizeof(Particle));
	hipMalloc(&deviceData_dev, sizeof(DeviceData));

	hipMemcpy(
		deviceData_host.read,
		writeParticles,
		particleCount * sizeof(Particle),
		hipMemcpyHostToDevice
	);
	hipMemcpy(
		deviceData_host.write,
		writeParticles,
		particleCount * sizeof(Particle),
		hipMemcpyHostToDevice
	);
	hipMemcpy(
		deviceData_dev,
		&deviceData_host, 
		sizeof(DeviceData),
		hipMemcpyHostToDevice
	);
		
	const size_t minStepsPerFrame = 64;
physicsThreadLoop:
	for(size_t i = 0; i < minStepsPerFrame || rendering; ++i) {
		// CUDA kernel
		physicsKernel<<<particleCount >> 2,dim3(4,8)>>>(deviceData_dev);

		// Swap device buffers
		Particle* tempParticleBuffer = deviceData_host.read;
		deviceData_host.read = deviceData_host.write;
		deviceData_host.write = tempParticleBuffer;
		hipMemcpy(
			deviceData_dev,
			&deviceData_host, 
			sizeof(DeviceData),
			hipMemcpyHostToDevice
		);

	}

	// Copy memory device to host
	hipMemcpy(
		writeParticles,
		deviceData_host.read,
		particleCount * sizeof(Particle),
		hipMemcpyDeviceToHost
	);

	// Swap host buffers
	Particle* tempParticles = writeParticles;
	writeParticles = readParticles;
	readParticles = tempParticles;
	rendering = true;

	// Handoff to render thread
	goto physicsThreadLoop;
	pthread_exit(NULL);
	return NULL;
}
