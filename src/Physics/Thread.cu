#include <Cloth/Physics.cuh>
#include <Cloth/Helper.cuh>
#include <stdio.h>

void* physicsThreadFunc(void* nothing) {
	// Initialize CUDA context
	DeviceData deviceData_host = {
		.particleCount = particleCount,
		.read = NULL,
		.write = NULL
	};
	DeviceData* deviceData_dev;

	hipMalloc(&deviceData_host.read, particleCount * sizeof(Particle));
	hipMalloc(&deviceData_host.write, particleCount * sizeof(Particle));
	hipMalloc(&deviceData_dev, sizeof(DeviceData));

	hipMemcpy(
		deviceData_host.read,
		writeParticles,
		particleCount * sizeof(Particle),
		hipMemcpyHostToDevice
	);
	hipMemcpy(
		deviceData_host.write,
		writeParticles,
		particleCount * sizeof(Particle),
		hipMemcpyHostToDevice
	);
	hipMemcpy(
		deviceData_dev,
		&deviceData_host, 
		sizeof(DeviceData),
		hipMemcpyHostToDevice
	);

	const size_t minStepsPerFrame = 64;
physicsThreadLoop:
	for(size_t i = 0; i < minStepsPerFrame || rendering; ++i) {
		
		// Instantiate struct with input data
		InputData inputData = InputData(GraphicsState::isClicking,
										GraphicsState::wasClickedLastFrame,
										GraphicsState::mouseY,
										GraphicsState::mouseZ);


		// CUDA kernel
		physicsKernel<<<(particleCount >> 2),(8 << 2)>>>(deviceData_dev, inputData);

		// Swap device buffers
		Particle* tempParticleBuffer = deviceData_host.read;
		deviceData_host.read = deviceData_host.write;
		deviceData_host.write = tempParticleBuffer;
		hipMemcpy(
			deviceData_dev,
			&deviceData_host, 
			sizeof(DeviceData),
			hipMemcpyHostToDevice
		);

	}

	// Copy memory device to host
	hipMemcpy(
		writeParticles,
		deviceData_host.read,
		particleCount * sizeof(Particle),
		hipMemcpyDeviceToHost
	);

	// Swap host buffers
	Particle* tempParticles = writeParticles;
	writeParticles = readParticles;
	readParticles = tempParticles;
	rendering = true;

	// Handoff to render thread
	goto physicsThreadLoop;
	pthread_exit(NULL);
	return NULL;
}
