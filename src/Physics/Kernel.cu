#include "hip/hip_runtime.h"
#include <Cloth/Physics.cuh>
#include <Cloth/Helper.cuh>

__global__
void physicsKernel(DeviceData* devData, InputData inputData) {
	__shared__ Particle particles[4];
	__shared__ Vector4 bindingPositions[4][8];

	// BlockDim.x = 4

	int blockParticleIndex = threadIdx.x >> 3; // [0,4)
	int intraParticleIndex = threadIdx.x & 0b111; // [0,8)
	int globalParticleIndex = (blockIdx.x << 2) + blockParticleIndex;
	int globalBindingIndex = 0;

	//------------------------------------//
	// Read global memory
	//------------------------------------//

	// Read particles from global memory
	if(!intraParticleIndex) {
		particles[blockParticleIndex] = devData->read[globalParticleIndex];

		// Gravity
		particles[blockParticleIndex].velocity.x[1] += -10.0 * TIME_STEP;
	}
	
	// Add force from mouse click
	if (inputData.isClicking) {
			particles[blockParticleIndex].velocity.x[1] += 
				inputData.mouseY * TIME_STEP;
			particles[blockParticleIndex].velocity.x[2] += 
				inputData.mouseZ * TIME_STEP;
	}


	// Read bindings from global memory
	globalBindingIndex =
		particles[blockParticleIndex].bindings[intraParticleIndex].index;
	bindingPositions[blockParticleIndex][intraParticleIndex] =
		devData->read[globalBindingIndex].position;

	//------------------------------------//
	// Calculation
	//------------------------------------//

	bindingForces(particles, bindingPositions);

	//------------------------------------//
	// Write global memory
	//------------------------------------//

	// Update particle position and write to global memory
	if(!intraParticleIndex) {
		// write back data
		devData->write[globalParticleIndex] = particles[blockParticleIndex];
	}
}
