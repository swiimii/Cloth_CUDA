#include "Physics.cuh"
#include <stdio.h>

void* physicsThreadFunc(void* nothing) {
	// Initialize CUDA context
	DeviceData deviceData_host = {
		.particleCount = particleCount,
		.read = NULL,
		.write = NULL
	};
	DeviceData* deviceData_dev;

	hipMalloc(&deviceData_host.read, particleCount * sizeof(Particle));
	hipMalloc(&deviceData_host.write, particleCount * sizeof(Particle));
	hipMalloc(&deviceData_dev, sizeof(DeviceData));

	hipMemcpy(
		deviceData_host.read,
		particleBuffer,
		particleCount * sizeof(Particle),
		hipMemcpyHostToDevice
	);
	hipMemcpy(
		deviceData_host.write,
		particleBuffer,
		particleCount * sizeof(Particle),
		hipMemcpyHostToDevice
	);
	hipMemcpy(
		deviceData_dev,
		&deviceData_host, 
		sizeof(DeviceData),
		hipMemcpyHostToDevice
	);
		
	const size_t stepsPerFrame = 25;
physicsThreadLoop:
	for(size_t i = 0; i < stepsPerFrame || rendering; ++i) {
		// CUDA kernel
		physicsKernel<<<particleCount >> 2,dim3(4,8)>>>(deviceData_dev);

		// Swap device buffers
		Particle* tempParticleBuffer = deviceData_host.read;
		deviceData_host.read = deviceData_host.write;
		deviceData_host.write = tempParticleBuffer;
		hipMemcpy(
			deviceData_dev,
			&deviceData_host, 
			sizeof(DeviceData),
			hipMemcpyHostToDevice
		);

	}

	//fprintf(stderr,"[P] Copy dev to host\n");

	// Copy memory device to host
	hipMemcpy(
		particleBuffer,
		deviceData_host.read,
		particleCount * sizeof(Particle),
		hipMemcpyDeviceToHost
	);

	// Copy device buffer into host write buffer
	for(size_t i = 0; i < particleCount; ++i)
		writePositions[i] = particleBuffer[i].position;

	// Swap host buffers
	Vector4* tempPositions = writePositions;
	writePositions = readPositions;
	readPositions = tempPositions;
	rendering = true;

	// Handoff to render thread
	goto physicsThreadLoop;
	pthread_exit(NULL);
	return NULL;
}
